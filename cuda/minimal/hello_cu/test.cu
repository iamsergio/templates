
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello()
{
    printf("Hello World from GPU!%d\n", threadIdx.x);
}

void my_kernel_wrapper()
{
    cuda_hello<<<1, 1>>>();
}
